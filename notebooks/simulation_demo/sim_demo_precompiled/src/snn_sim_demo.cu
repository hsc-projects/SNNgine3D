#include "hip/hip_runtime.h"
#include <snn_sim_demo.cuh>


LaunchParameters::LaunchParameters()
{
	block_size = 0;
	grid_size = 0;
	block3 = dim3(block_size);
	grid3 = dim3(grid_size);
}

LaunchParameters::LaunchParameters(const int n_threads_x, void*init_func)
{
	func = init_func;
	init_sizes(n_threads_x, init_func);
	block3 = dim3(block_size);
	grid3 = dim3(grid_size);
}


void LaunchParameters::init_sizes(const int n_threads_x, void* init_func)
{	
	hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, func, 0, 64);
	grid_size = (n_threads_x + block_size - 1) / block_size;
	if (grid_size == 1) {
		block_size = std::min(block_size, n_threads_x);
	}
}



SynapticCurrentUpdater::SynapticCurrentUpdater(
    const int N_,
    const int S_,
    const int D_,
    const int T_,

    int* N_rep_, 
    int* N_delays_, 
    
    int* N_types_, 
    float* N_states_,
	float* N_weights_,
	
    float* fired_,
	float* firing_times_,
	int* firing_idcs_,
	int* firing_counts_
){
    
	N = N_;
	S = S_;
	D = D_;
    T = T_;

    N_rep = N_rep_;
    N_delays = N_delays_;

	N_types = N_types_;
    N_states = N_states_;
	N_weights = N_weights_;

	// Pointer initializations
	fired = fired_;	
	firing_times = firing_times_;
	firing_idcs = firing_idcs_;
	firing_counts = firing_counts_;

	// Initially, all pointers point to the start of the respective array.
	firing_times_write = firing_times;
	firing_times_read = firing_times;

	firing_idcs_write = firing_idcs;
	firing_idcs_read = firing_idcs;
	
	firing_counts_write = firing_counts;


	reset_firing_times_ptr_threshold = 13 * N;
	reset_firing_count_idx_threshold = 2 * T;

	// Cusparse Initialization (must only be done once)
	checkCusparseErrors(hipsparseCreate(&fired_handle));
	checkCusparseErrors(hipsparseCreateDnMat(&firing_times_dense,
		1, N, N,
		fired,
		HIP_R_32F, HIPSPARSE_ORDER_ROW));
	
	checkCusparseErrors(hipsparseCreateCsr(&firing_times_sparse, 1, N, 0,
		firing_counts_write,
		firing_idcs_write,
		firing_times_write,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

	checkCusparseErrors(hipsparseDenseToSparse_bufferSize(
		fired_handle, firing_times_dense, firing_times_sparse,
		HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
		&fired_buffer_size));
	checkCudaErrors(hipMalloc(&fired_buffer, fired_buffer_size));

}


__global__ void update_current_(
	const int N, const int S, const int D,
	const int* fired_idcs_read, const int* fired_idcs, 
	const float* firing_times_read, const float* firing_times,
	const int* N_flags, const int* N_rep, float* N_weights, float* N_states, const int* N_delays,
	const int n_fired_m1_to_end, const int n_fired,
	const int t
)
{
	const int fired_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (fired_idx < n_fired)
	{
		int n;  			// pre-synaptic neuron
		int firing_time;	// firing time of the pre-synaptic neuron

		if (fired_idx < n_fired_m1_to_end)
		{
			// global index of firing-array < len(fired-array) 
			// -> use the trailing pointer
			n = fired_idcs_read[fired_idx];
			firing_time = __float2int_rn(firing_times_read[fired_idx]);
		}
		else
		{
			// global index of firing-array >= len(fired-array) 
			// -> use the 'normal' pointer
			n = fired_idcs[fired_idx - n_fired_m1_to_end];
			firing_time = __float2int_rn(firing_times[fired_idx - n_fired_m1_to_end]);
		}

		int delay = t - firing_time;  // time passed since the neuron fired
		const int delay_idx = n + N * (delay);  

		int snk_N; 		// post-synaptic Neuron-ID
		int idx;		// synapse-index 
		
		// row-index of the first synapse with a delay d_next = delay + 1
		int s_end = N_delays[delay_idx + N];  

		float w;		// weight of the synapse

		// loop through all synapses with a delay d == delay
		for (int s = N_delays[delay_idx]; s < s_end; s++)
		{
			idx = n + N * s;		// synapse-index 
			snk_N = N_rep[idx];		// post-synaptic Neuron-ID

			if (snk_N >= 0)  // allows to delete synapses by placing -1
			{
				// add the weight of synapse to the current-value of the post-synaptic neuron
				w  =  N_weights[idx];			
				atomicAdd(&N_states[snk_N + 7 * N], w);		
			}
		}
	}
	
}


void SynapticCurrentUpdater::print_fired(){
	// TODO: rewrite the copying + printing
	printf("fired         = [");
	for (int i = 0; i < N; i++) {
		float fired_value;
		hipMemcpy(&fired_value, fired + i, 
			sizeof(float), hipMemcpyDeviceToHost);
		printf("%.0f", fired_value);
		if (i < N - 1){
			printf(", ");
		}

	}
	printf("].\n");
}


void SynapticCurrentUpdater::print_info(bool print_counts){
	
	if (print_counts){
		printf("t = %d,", t);
		printf("\nn_fired                      = %d, ", n_fired);
		if (n_fired < 10) printf(" ");
		printf("n_fired_m1_to_end            = %d,", n_fired_m1_to_end);
		printf("\nn_fired_0                    = %d, ", n_fired_0);
		if (n_fired_0 < 10) printf(" ");
		printf("n_fired_m1                   = %d,", n_fired_m1);
		printf("\nn_fired_total                = %d, ", n_fired_total);
		if (n_fired_total < 10) printf(" ");
		printf("n_fired_total_m1             = %d,", n_fired_total_m1);
		// printf("\nfiring_counts_write=%p", (void * )firing_counts_write);
		printf("\nfiring_counts_write (offset) = %ld,", firing_counts_write - firing_counts);
		printf("\nfiring_idcs_read    (offset) = %ld, ", firing_idcs_read - firing_idcs);
		if (firing_idcs_read - firing_idcs < 10) printf(" ");
		printf("firing_idcs_write  (offset)  = %ld,", firing_idcs_write - firing_idcs);
		printf("\nfiring_times_read   (offset) = %ld, ", firing_times_read - firing_times);
		if (firing_times_read - firing_times < 10) printf(" ");
		printf("firing_times_write (offset)  = %ld.", firing_times_write - firing_times);
		printf("\n");
	}
}


void SynapticCurrentUpdater::shift_sim_pointers(){

	checkCudaErrors(hipMemcpy(
		&n_fired_0, firing_counts + firing_counts_idx, sizeof(int), hipMemcpyDeviceToHost));

	n_fired_total += n_fired_0;
	n_fired += n_fired_0;
	firing_counts_idx += 2;

	if (n_fired_total > n_fired_total_m1) {
		n_fired_m1_to_end += n_fired_0;
	}


	if (t >= D)
	{
		hipMemcpy(&n_fired_m1, firing_counts + firing_counts_idx_m1, 
                   sizeof(int), hipMemcpyDeviceToHost);

		n_fired_total_m1 += n_fired_m1;
		n_fired -= n_fired_m1;
		n_fired_m1_to_end -= n_fired_m1;
		firing_counts_idx_m1 += 2;
	}

	if (n_fired_total <= reset_firing_times_ptr_threshold)
	{
		firing_times_write += n_fired_0;
		firing_idcs_write += n_fired_0;
	}
	else
	{
		firing_times_write = firing_times;
		firing_idcs_write = firing_idcs;
		n_fired_total = 0;
		resetting = true;
	}

	if (firing_counts_idx > reset_firing_count_idx_threshold){
		firing_counts_idx = 1;
		firing_counts_write = firing_counts;
	} else {
		firing_counts_write += 2;
	}
	
	if (firing_counts_idx_m1 > reset_firing_count_idx_threshold){
		firing_counts_idx_m1 = 1;	
	} 


	if (n_fired_total_m1 <= reset_firing_times_ptr_threshold)
	{
		firing_times_read += n_fired_m1;
		firing_idcs_read += n_fired_m1;
	}
	else
	{
		firing_times_read = firing_times;
		firing_idcs_read = firing_idcs;
		n_fired_m1_to_end = n_fired_total;
		n_fired_total_m1 = 0;
		resetting = false;
	}

	hipsparseCsrSetPointers(firing_times_sparse,
		firing_counts_write,
		firing_idcs_write,
		firing_times_write);

	checkCudaErrors(hipDeviceSynchronize());
}

void SynapticCurrentUpdater::dense_to_sparse_conversion(const bool verbose)
{	

	checkCudaErrors(hipDeviceSynchronize());

	// 2. DenseToSparse Conversion (using "write"-pointers).
	checkCusparseErrors(hipsparseDenseToSparse_analysis(
		fired_handle, firing_times_dense, firing_times_sparse,
		HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, fired_buffer));
	
	checkCudaErrors(hipDeviceSynchronize());

	checkCusparseErrors(hipsparseDenseToSparse_convert(
		fired_handle, firing_times_dense, firing_times_sparse,
		HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, fired_buffer));
	
	if (verbose) print_info(true);
	print_fired();
	
}

void SynapticCurrentUpdater::update_synaptic_current()
{	
	
	int block_dim_x = 32;
	int grid_dim_x = static_cast<int>(::ceilf(static_cast<float>(n_fired) 
									  / static_cast<float>(block_dim_x)));
	// 4. Synaptic current update (kernel; using "read"-pointers).
	update_current_ KERNEL_ARGS2(grid_dim_x, block_dim_x)(
		N, S, D,
		firing_idcs_read, firing_idcs,
		firing_times_read, firing_times,
		N_types, N_rep, N_weights, N_states, N_delays,
		n_fired_m1_to_end, n_fired, t
    );
	
	checkCudaErrors(hipDeviceSynchronize());

	t++;
}
